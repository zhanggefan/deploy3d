#include "hip/hip_runtime.h"
#include "common/cub.cuh"
#include "common/hash32.h"
#include "common/launch.cuh"
#include "common/macros.h"
#include "common/refnd.h"
#include "geometry.h"

using utils::GPU;
using utils::launch::CUDA_NUM_THREADS;
using utils::launch::getBlocks;
using utils::launch::KernelLoopX;
using utils::nd::Ref1D;
using utils::nd::Ref2D;
using utils::nd::Ref3D;
using utils::nd::RefND;
using utils::nd::Size;
using utils::nd::Vec;

namespace spconv {

static constexpr int TBSegment = 128;

template <class T> using mkS = typename std::make_signed_t<T>;
template <class T> using mkU = typename std::make_unsigned_t<T>;

namespace kernel {

template <size_t NDim, class Index> class ConvHashWriter {
 public:
  using HashTable = hash::LinearHashTable<Index, Index>;
  DEVICE_INLINE
  ConvHashWriter(Ref2D<mkU<Index>>& hashOut, HashTable& hash, const Size<NDim + 1>& outSpatialShape)
      : hashOut(hashOut), hash(hash), outSpatialShape(outSpatialShape) {}
  DEVICE_INLINE
  void operator()(size_t inIndex, size_t kernelOffset, Index* outCoor) {
    if (outSpatialShape.is_valid(outCoor)) {
      Index outSpatialOffset = outSpatialShape.offset(outCoor);
      hashOut(kernelOffset, inIndex) = hash.insert(outSpatialOffset, 1);
    }
  }

 private:
  Ref2D<mkU<Index>>& hashOut;
  HashTable& hash;
  const Size<NDim + 1>& outSpatialShape;
};

template <size_t NDim, class Index> class SubMHashWriter {
 public:
  using HashTable = hash::LinearHashTable<Index, Index>;
  DEVICE_INLINE
  SubMHashWriter(Ref2D<mkU<Index>>& hashOut, const HashTable& hash, const Size<NDim + 1>& outSpatialShape)
      : hashOut(hashOut), hash(hash), outSpatialShape(outSpatialShape) {}
  DEVICE_INLINE
  void operator()(size_t inIndex, size_t kernelOffset, Index* outCoor) {
    if (outSpatialShape.is_valid(outCoor)) {
      Index outSpatialOffset = outSpatialShape.offset(outCoor);
      Index outIndex;
      if (hash.lookup(outSpatialOffset, outIndex)) { hashOut(kernelOffset, inIndex) = outIndex; }
    }
  }

 private:
  Ref2D<mkU<Index>>& hashOut;
  const HashTable& hash;
  const Size<NDim + 1>& outSpatialShape;
};

template <size_t NDim, class Index>
__global__ void subMGeometryKernel(Ref2D<mkU<Index>> hashOut,
                                   const typename SubMHashWriter<NDim, Index>::HashTable hash,
                                   const Ref2D<Index> coorsIn,
                                   const Vec<NDim, Index> kernelSize,
                                   const Vec<NDim, Index> stride,
                                   const Vec<NDim, Index> padding,
                                   const Vec<NDim, Index> dilation,
                                   const Size<NDim + 1> outSpatialShape) {
  auto numActIn = coorsIn.size(0);
  SubMHashWriter<NDim, Index> writer(hashOut, hash, outSpatialShape);
  for (size_t ix : KernelLoopX(numActIn)) {
    detail::geometry::getOutPosLoop<NDim, Index, SubMHashWriter<NDim, Index>>(ix, coorsIn, kernelSize, stride, padding,
                                                                              dilation, writer, false);
  }
}

template <size_t NDim, class Index>
__global__ void convGeometryKernel(Ref2D<mkU<Index>> hashOut,
                                   typename ConvHashWriter<NDim, Index>::HashTable hash,
                                   const Ref2D<Index> coorsIn,
                                   const Vec<NDim, Index> kernelSize,
                                   const Vec<NDim, Index> stride,
                                   const Vec<NDim, Index> padding,
                                   const Vec<NDim, Index> dilation,
                                   const Size<NDim + 1> outSpatialShape,
                                   const bool transpose) {
  auto numActIn = coorsIn.size(0);
  ConvHashWriter<NDim, Index> writer(hashOut, hash, outSpatialShape);
  for (size_t ix : KernelLoopX(numActIn)) {
    detail::geometry::getOutPosLoop<NDim, Index, ConvHashWriter<NDim, Index>>(ix, coorsIn, kernelSize, stride, padding,
                                                                              dilation, writer, transpose);
  }
}

template <size_t NDim, class Index>
__global__ void coorsOutOrganizeKernel(Ref2D<Index> coorsOut,
                                       Index* numActOut,
                                       const typename kernel::ConvHashWriter<NDim, Index>::HashTable hash,
                                       const Ref1D<Index> uniqueIndex,
                                       const Size<NDim + 1> outSpatialShape,
                                       const Index numSample) {
  auto numElem = hash.size();
  for (size_t ix : KernelLoopX(hash.size())) {
    auto spatialOffset = hash.keys()[ix];
    Index outIndex = uniqueIndex[ix];
    if (spatialOffset != hash.EMPTY && outIndex < numSample) {
      outSpatialShape.deserialize(&coorsOut(outIndex, 0), spatialOffset);
      outIndex++;
    }
    if (ix == numElem - 1) *numActOut = min(outIndex, numSample);
  }
}

template <class Index>
__global__ void indexOrganizeKernel(Ref1D<Index> GatherIn,
                                    Ref1D<Index> ScatterOut,
                                    Ref1D<Index> KernelOffset,
                                    Index* numIndexPtr,
                                    const Ref2D<Index> outputIndex,
                                    const Ref1D<Index> outputIndexPadCumSum,
                                    const Ref2D<mkU<Index>> hashOut) {
  auto numActIn = hashOut.size(1);
  for (size_t ix : KernelLoopX(hashOut.numel())) {
    auto scatterTo = static_cast<Index>(hashOut[ix]);
    if (scatterTo >= 0) {
      auto outIndex = outputIndex[ix] + outputIndexPadCumSum[ix / numActIn];
      GatherIn[outIndex] = ix % numActIn;
      KernelOffset[outIndex] = ix / numActIn;
      ScatterOut[outIndex] = scatterTo;
    }
    if (ix == hashOut.numel() - 1) {
      auto outIndex = outputIndex[ix] + outputIndexPadCumSum[ix / numActIn];
      outIndex += scatterTo >= 0 ? 1 : 0;
      *numIndexPtr = outIndex + ((TBSegment - (outIndex & (TBSegment - 1))) & (TBSegment - 1));
    }
  }
}

template <class Index>
__global__ void outputIndexPadOrganizeKernel(Ref1D<Index> outputIndexPad, const Ref2D<Index> outputIndex) {
  auto kVol = outputIndex.size(0);
  auto numActIn = outputIndex.size(1);
  for (size_t ix : KernelLoopX(kVol)) {
    Index numActKernel = ix ? outputIndex(ix, 0) - outputIndex(ix - 1, 0) : 0;
    outputIndexPad[ix] = ((TBSegment - (numActKernel & (TBSegment - 1))) & (TBSegment - 1));
  }
}

template <class Index>
__global__ void hashOutIndexKernel(Ref2D<mkU<Index>> hashOut, const Ref1D<Index> uniqueIndex, const Index numSample) {
  size_t numElem = hashOut.numel();
  for (size_t ix : KernelLoopX(numElem)) {
    auto slot = hashOut[ix];
    if (slot < uniqueIndex.numel()) {
      auto uid = uniqueIndex[slot];
      if (uid < numSample) {
        hashOut[ix] = uid;
      } else {
        hashOut[ix] = ~mkU<Index>(0);
      }
    }
  }
}

template <size_t NDim, class Index>
__global__ void setSubMHashKernel(typename SubMHashWriter<NDim, Index>::HashTable hash,
                                  const Ref2D<Index> coorsIn,
                                  const Size<NDim + 1> outSpatialShape) {
  for (size_t ix : KernelLoopX(coorsIn.size(0))) {
    Index index = outSpatialShape.offset(&coorsIn(ix, 0));
    hash.insert(index, ix);
  }
}

template <class Hash> __global__ void resetHashKernel(Hash hash) {
  for (size_t ix : KernelLoopX(hash.size())) { hash.keys()[ix] = hash.EMPTY; }
}

}  // namespace kernel

namespace func {

constexpr int hashSpace = 2;

template <class Index> HOST_DEVICE_INLINE Index FF() {
  const mkU<Index> ff = ~mkU<Index>(0);
  return reinterpret_cast<const Index&>(ff);
}

struct Valid {
  template <class Index> DEVICE_INLINE bool operator()(const Index& a) const { return a != FF<Index>(); }
};

struct Valid_Counter {
  template <class Index> DEVICE_INLINE Index operator()(const Index& a) const {
    return a != FF<Index>() ? Index(1) : Index(0);
  }
};

size_t getMinPower2(size_t x) {
  size_t y;
  for (y = 1; y < x; y = y << 1) {}
  return y;
}

template <size_t NDim, class Index>
size_t createSparseConvIndexMalloc(const GPU& d,
                                   const size_t numInput,
                                   const Vec<NDim, Index>& kernelSize,
                                   const bool oneTimeMalloc) {
  size_t reqBytes_, reqBytes = 0;
  size_t kVol = 1;
#pragma unroll
  for (size_t i = 0; i < NDim; i++) { kVol *= kernelSize[i]; }
  size_t outElemNumMaxPower2 = getMinPower2(numInput * kVol);
  size_t numElemHash = getMinPower2(numInput * kVol * hashSpace);

  hipError_t status;

  for (size_t scale = oneTimeMalloc ? 1 : outElemNumMaxPower2; scale <= outElemNumMaxPower2; scale = scale << 1) {
    status = DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum<
        DEPLOY3D_CUB_NS_QUALIFIER::hipcub::TransformInputIterator<Index, Valid_Counter, mkU<Index>*>, Index*>(
        nullptr, reqBytes_, {nullptr, {}}, nullptr, scale, d.getStream());
    CHECK_RETURN_STATUS(status);
    reqBytes = reqBytes > reqBytes_ ? reqBytes : reqBytes_;
  }

  for (size_t scale = oneTimeMalloc ? 1 : numElemHash; scale <= numElemHash; scale = scale << 1) {
    status = DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum<Index*, Index*>(
        nullptr, reqBytes_, nullptr, nullptr, numElemHash, d.getStream());
    CHECK_RETURN_STATUS(status);
    reqBytes = reqBytes > reqBytes_ ? reqBytes : reqBytes_;
  }

  status = DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::InclusiveSum<Index*, Index*>(nullptr, reqBytes_, nullptr,
                                                                                    nullptr, kVol, d.getStream());
  CHECK_RETURN_STATUS(status);
  reqBytes = reqBytes > reqBytes_ ? reqBytes : reqBytes_;

  reqBytes += (3 * numElemHash + 4 * outElemNumMaxPower2) * sizeof(Index);
  return reqBytes;
}

template <size_t NDim, class Index>
void createSparseConvIndex(const GPU& d,
                           Ref1D<uint8_t>& workingStorage,
                           Ref1D<Index>& GatherIn,
                           Ref1D<Index>& ScatterOut,
                           Ref1D<Index>& KernelOffset,
                           Ref2D<Index>& coorsOut,
                           Index* numIndexPtr,
                           Index* numOutPtr,
                           const Ref2D<Index>& coorsIn,
                           const Vec<NDim, Index>& kernelSize,
                           const Vec<NDim, Index>& stride,
                           const Vec<NDim, Index>& padding,
                           const Vec<NDim, Index>& dilation,
                           const Size<NDim + 1>& outSpatialShape,
                           const bool transpose,
                           const Index numSample) {
  Index numActIn = coorsIn.size(0);
  if (numActIn == 0) {
    hipMemsetAsync(numIndexPtr, 0, sizeof(Index), d.getStream());
    hipMemsetAsync(numOutPtr, 0, sizeof(Index), d.getStream());
    return;
  }

  ssize_t kVol = 1;
#pragma unroll
  for (size_t i = 0; i < NDim; i++) { kVol *= kernelSize[i]; }
  ssize_t outElemNumMaxPower2 = getMinPower2(kVol * numActIn);
  ssize_t numElemHash = getMinPower2(kVol * numActIn * hashSpace);

  uint8_t* workingStoragePtr = workingStorage.data();

  Ref1D<Index> hashKeys(reinterpret_cast<Index*>(workingStoragePtr), {numElemHash});
  workingStoragePtr += hashKeys.numby();
  Ref1D<Index> hashValues(reinterpret_cast<Index*>(workingStoragePtr), {numElemHash});
  workingStoragePtr += hashValues.numby();
  Ref1D<Index> uniqueIndex(reinterpret_cast<Index*>(workingStoragePtr), {numElemHash});
  workingStoragePtr += uniqueIndex.numby();

  typename kernel::ConvHashWriter<NDim, Index>::HashTable hash(hashKeys.data(), hashValues.data(), numElemHash);

  Ref2D<mkU<Index>> hashOut(reinterpret_cast<mkU<Index>*>(workingStoragePtr), {kVol, numActIn});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(mkU<Index>);

  Ref2D<Index> outputIndex(reinterpret_cast<Index*>(workingStoragePtr), {kVol, numActIn});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  Ref1D<Index> outputIndexPad(reinterpret_cast<Index*>(workingStoragePtr), {kVol});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  Ref1D<Index> outputIndexPadCumSum(reinterpret_cast<Index*>(workingStoragePtr), {kVol});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  size_t workingStorageBytes = workingStorage.size(0) - (workingStoragePtr - workingStorage.data());

  {  // conv I-O geometry
    hipMemsetAsync(hashOut.data(), 0xFF, outElemNumMaxPower2 * sizeof(mkU<Index>), d.getStream());
    hipMemsetAsync(hashValues.data(), 0x00, hashValues.numby(), d.getStream());
    kernel::resetHashKernel<typename kernel::ConvHashWriter<NDim, Index>::HashTable>
        <<<getBlocks(hash.size()), CUDA_NUM_THREADS, 0, d.getStream()>>>(hash);
    kernel::convGeometryKernel<NDim, Index><<<getBlocks(numActIn), CUDA_NUM_THREADS, 0, d.getStream()>>>(
        hashOut, hash, coorsIn, kernelSize, stride, padding, dilation, outSpatialShape, transpose);
    CHECK_CUDA_ERR();
  }

  {  // make output unique index
    // input:
    //   hashOut: geometric io lookup table:
    //      ({input idx}, {filter idx})
    //      -> {output coors hash slot} or -1 (output invalid)
    //
    // output:
    //   hashOut: geometric io lookup table:
    //      ({input idx}, {filter idx})
    //      -> {output idx}
    //   numOutPtr: number of output coors
    //   coorsOut: output coors
    DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum<Index*, Index*>(
        workingStoragePtr, workingStorageBytes, hashValues.data(), uniqueIndex.data(), hash.size(), d.getStream());
    kernel::coorsOutOrganizeKernel<NDim, Index><<<getBlocks(hash.size()), CUDA_NUM_THREADS, 0, d.getStream()>>>(
        coorsOut, numOutPtr, hash, uniqueIndex, outSpatialShape, numSample);
    kernel::hashOutIndexKernel<Index>
        <<<getBlocks(hashOut.numel()), CUDA_NUM_THREADS, 0, d.getStream()>>>(hashOut, uniqueIndex, numSample);
    CHECK_CUDA_ERR();
  }

  {  // organize output
    // input:
    //   hashOut: geometric io lookup table:
    //      ({input idx}, {filter idx})
    //      -> {output idx} or -1 (output invalid)
    //
    // output:
    //   numIndexPtr: length of GatherIn/ScatterOut/KernelOffset
    //   GatherIn: index conv gather source. calculate by filtering out -1
    //             from hashOut and then decoding the input index
    //   ScatterOut: index conv scatter target. calculate by filtering out -1
    //               from hashOut and then looking up the hashOut
    //   kernelOffset: index conv kernel offset index
    hipMemsetAsync(GatherIn.data(), 0xFF, GatherIn.numby(), d.getStream());
    hipMemsetAsync(ScatterOut.data(), 0xFF, ScatterOut.numby(), d.getStream());
    hipMemsetAsync(KernelOffset.data(), 0xFF, KernelOffset.numby(), d.getStream());
    DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum(
        workingStoragePtr, workingStorageBytes,
        DEPLOY3D_CUB_NS_QUALIFIER::hipcub::TransformInputIterator<Index, Valid_Counter, mkU<Index>*>(hashOut.data(), {}),
        outputIndex.data(), outElemNumMaxPower2, d.getStream());
    kernel::outputIndexPadOrganizeKernel<Index>
        <<<getBlocks(kVol), CUDA_NUM_THREADS, 0, d.getStream()>>>(outputIndexPad, outputIndex);
    DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::InclusiveSum<Index*, Index*>(
        workingStoragePtr, workingStorageBytes, outputIndexPad.data(), outputIndexPadCumSum.data(), kVol,
        d.getStream());
    kernel::indexOrganizeKernel<Index><<<getBlocks(hashOut.numel()), CUDA_NUM_THREADS, 0, d.getStream()>>>(
        GatherIn, ScatterOut, KernelOffset, numIndexPtr, outputIndex, outputIndexPadCumSum, hashOut);
    CHECK_CUDA_ERR();
  }
}

template <size_t NDim, class Index>
size_t createSparseSubMIndexMalloc(const GPU& d,
                                   const size_t numInput,
                                   const Vec<NDim, Index>& kernelSize,
                                   const bool oneTimeMalloc) {
  size_t reqBytes_, reqBytes = 0;
  size_t kVol = 1;
#pragma unroll
  for (size_t i = 0; i < NDim; i++) { kVol *= kernelSize[i]; }
  size_t outElemNumMaxPower2 = getMinPower2(numInput * kVol);
  size_t numElemHash = getMinPower2(numInput * hashSpace);

  hipError_t status;
  for (size_t scale = oneTimeMalloc ? 1 : outElemNumMaxPower2; scale <= outElemNumMaxPower2; scale = scale << 1) {
    status = DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum<
        DEPLOY3D_CUB_NS_QUALIFIER::hipcub::TransformInputIterator<Index, Valid_Counter, mkU<Index>*>, Index*>(
        nullptr, reqBytes_, {nullptr, {}}, nullptr, scale, d.getStream());
    CHECK_RETURN_STATUS(status);
    reqBytes = reqBytes > reqBytes_ ? reqBytes : reqBytes_;
  }

  status = DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::InclusiveSum<Index*, Index*>(nullptr, reqBytes_, nullptr,
                                                                                    nullptr, kVol, d.getStream());
  CHECK_RETURN_STATUS(status);
  reqBytes = reqBytes > reqBytes_ ? reqBytes : reqBytes_;

  reqBytes += (2 * numElemHash + 4 * outElemNumMaxPower2) * sizeof(Index);
  return reqBytes;
}

template <size_t NDim, class Index>
void createSparseSubMIndex(const GPU& d,
                           Ref1D<uint8_t>& workingStorage,
                           Ref1D<Index>& GatherIn,
                           Ref1D<Index>& ScatterOut,
                           Ref1D<Index>& KernelOffset,
                           Index* numIndexPtr,
                           const Ref2D<Index>& coorsIn,
                           const Vec<NDim, Index>& kernelSize,
                           const Vec<NDim, Index>& stride,
                           const Vec<NDim, Index>& padding,
                           const Vec<NDim, Index>& dilation,
                           const Size<NDim + 1>& outSpatialShape) {
  Index numActIn = coorsIn.size(0);
  if (numActIn == 0) {
    hipMemsetAsync(numIndexPtr, 0, sizeof(Index), d.getStream());
    return;
  }

  ssize_t kVol = 1;
#pragma unroll
  for (size_t i = 0; i < NDim; i++) { kVol *= kernelSize[i]; }
  ssize_t outElemNumMaxPower2 = getMinPower2(kVol * numActIn);
  ssize_t numElemHash = getMinPower2(numActIn * hashSpace);

  uint8_t* workingStoragePtr = workingStorage.data();

  Ref1D<Index> hashKeys(reinterpret_cast<Index*>(workingStoragePtr), {numElemHash});
  workingStoragePtr += hashKeys.numby();
  Ref1D<Index> hashValues(reinterpret_cast<Index*>(workingStoragePtr), {numElemHash});
  workingStoragePtr += hashValues.numby();

  typename kernel::SubMHashWriter<NDim, Index>::HashTable hash(hashKeys.data(), hashValues.data(), numElemHash);

  Ref2D<mkU<Index>> hashOut(reinterpret_cast<mkU<Index>*>(workingStoragePtr), {kVol, numActIn});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(mkU<Index>);

  Ref2D<Index> outputIndex(reinterpret_cast<Index*>(workingStoragePtr), {kVol, numActIn});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  Ref1D<Index> outputIndexPad(reinterpret_cast<Index*>(workingStoragePtr), {kVol});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  Ref1D<Index> outputIndexPadCumSum(reinterpret_cast<Index*>(workingStoragePtr), {kVol});
  workingStoragePtr += outElemNumMaxPower2 * sizeof(Index);

  size_t workingStorageBytes = workingStorage.size(0) - (workingStoragePtr - workingStorage.data());

  {  // subM conv I-O geometry
    hipMemsetAsync(hashOut.data(), 0xFF, outElemNumMaxPower2 * sizeof(mkU<Index>), d.getStream());
    kernel::resetHashKernel<typename kernel::SubMHashWriter<NDim, Index>::HashTable>
        <<<getBlocks(hash.size()), CUDA_NUM_THREADS, 0, d.getStream()>>>(hash);
    kernel::setSubMHashKernel<NDim, Index>
        <<<getBlocks(numActIn), CUDA_NUM_THREADS, 0, d.getStream()>>>(hash, coorsIn, outSpatialShape);
    kernel::subMGeometryKernel<NDim, Index><<<getBlocks(numActIn), CUDA_NUM_THREADS, 0, d.getStream()>>>(
        hashOut, hash, coorsIn, kernelSize, stride, padding, dilation, outSpatialShape);
    CHECK_CUDA_ERR();
  }
  {  // make bufferFromIn and bufferOffset
    hipMemsetAsync(GatherIn.data(), 0xFF, GatherIn.numby(), d.getStream());
    hipMemsetAsync(ScatterOut.data(), 0xFF, ScatterOut.numby(), d.getStream());
    hipMemsetAsync(KernelOffset.data(), 0xFF, KernelOffset.numby(), d.getStream());
    DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::ExclusiveSum(
        workingStoragePtr, workingStorageBytes,
        DEPLOY3D_CUB_NS_QUALIFIER::hipcub::TransformInputIterator<Index, Valid_Counter, mkU<Index>*>(hashOut.data(), {}),
        outputIndex.data(), outElemNumMaxPower2, d.getStream());
    kernel::outputIndexPadOrganizeKernel<Index>
        <<<getBlocks(kVol), CUDA_NUM_THREADS, 0, d.getStream()>>>(outputIndexPad, outputIndex);
    DEPLOY3D_CUB_NS_QUALIFIER::hipcub::DeviceScan::InclusiveSum<Index*, Index*>(
        workingStoragePtr, workingStorageBytes, outputIndexPad.data(), outputIndexPadCumSum.data(), kVol,
        d.getStream());
    kernel::indexOrganizeKernel<Index><<<getBlocks(hashOut.numel()), CUDA_NUM_THREADS, 0, d.getStream()>>>(
        GatherIn, ScatterOut, KernelOffset, numIndexPtr, outputIndex, outputIndexPadCumSum, hashOut);
    CHECK_CUDA_ERR();
  }
  CHECK_CUDA_ERR();
};

}  // namespace func
}  // namespace spconv

#define SPECIALIZE(NDim, Index)                                                                                        \
  template size_t spconv::func::createSparseConvIndexMalloc<NDim, Index>(                                              \
      const GPU& d, const size_t numInput, const Vec<NDim, Index>& kernelSize, const bool oneTimeMalloc);              \
                                                                                                                       \
  template void spconv::func::createSparseConvIndex<NDim, Index>(                                                      \
      const GPU& d, Ref1D<uint8_t>& workingStorage, Ref1D<Index>& GatherIn, Ref1D<Index>& ScatterOut,                  \
      Ref1D<Index>& KernelOffset, Ref2D<Index>& coorsOut, Index* numIndexPtr, Index* numOutPtr,                        \
      const Ref2D<Index>& coorsIn, const Vec<NDim, Index>& kernelSize, const Vec<NDim, Index>& stride,                 \
      const Vec<NDim, Index>& padding, const Vec<NDim, Index>& dilation, const Size<NDim + 1>& outSpatialShape,        \
      const bool transpose, const Index numSample);                                                                    \
                                                                                                                       \
  template size_t spconv::func::createSparseSubMIndexMalloc<NDim, Index>(                                              \
      const GPU& d, const size_t numInput, const Vec<NDim, Index>& kernelSize, const bool oneTimeMalloc);              \
                                                                                                                       \
  template void spconv::func::createSparseSubMIndex<NDim, Index>(                                                      \
      const GPU& d, Ref1D<uint8_t>& workingStorage, Ref1D<Index>& GatherIn, Ref1D<Index>& ScatterOut,                  \
      Ref1D<Index>& KernelOffset, Index* numIndexPtr, const Ref2D<Index>& coorsIn, const Vec<NDim, Index>& kernelSize, \
      const Vec<NDim, Index>& stride, const Vec<NDim, Index>& padding, const Vec<NDim, Index>& dilation,               \
      const Size<NDim + 1>& outSpatialShape);

SPECIALIZE(1, int);
SPECIALIZE(2, int);
SPECIALIZE(3, int);
SPECIALIZE(4, int);